#include "hip/hip_runtime.h"
// Scalpel Copyright (C) 2005-8 by Golden G. Richard III and 
// 2007-8 by Vico Marziale.
// Written by Golden G. Richard III and Vico Marziale.
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License as
// published by the Free Software Foundation; either version 2 of the
// License, or (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA
// 02110-1301, USA.
//
// Thanks to Kris Kendall, Jesse Kornblum, et al for their work 
// on Foremost.  Foremost 0.69 was used as the starting point for 
// Scalpel, in 2005.
//


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <cutil.h> 
#include "common.h"


// returns TRUE if a is alpha
#define ISALPHA(a) ((a >= 'A' && a <= 'Z') || (a >= 'a' && a <= 'z'))


// Globals.
__constant__ char pattern[MAX_PATTERNS][MAX_PATTERN_LENGTH];
__constant__ char lookup_headers[LOOKUP_ROWS][LOOKUP_COLUMNS]; 
__constant__ char lookup_footers[LOOKUP_ROWS][LOOKUP_COLUMNS]; 

char host_patterns[MAX_PATTERNS][MAX_PATTERN_LENGTH];
char hostlookup_headers[LOOKUP_ROWS][LOOKUP_COLUMNS]; 
char hostlookup_footers[LOOKUP_ROWS][LOOKUP_COLUMNS]; 

int GPU_ERROR = 0;	// Unused for now, but shouldn't be ;) 

char *dev_in;		// device buffer for input
char *dev_out;	// device buffer for results output
int *dev_count;	// device buffer for intermediate results


// Forward declarations for private functions.
static void checkCUDAError(const char *msg);
/*
static int set_last_device();		// currently unused
static int enumerate_devices();	// currently unused
*/

// Performs search for headers and footers in bufize dev_in, and puts encoded
// results into dev_out. 
__global__ void gpudigbuffer_kernel(char *dev_in, char *dev_out, int bufsize,
									int longestneedle, int *dev_count, char wildcard) {

 // Per-block shared memory, size determined by kernel invocation.
  extern __shared__  char sdata[];
  
  // My thread, block id, the number of threads per block, and my global tid, 
  // aka my unique thread tid across all thread blocks.
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  const unsigned int num_thread_blocks = gridDim.x;
  const unsigned int g_tid = (THREADS_PER_BLOCK * bid) + tid; 
   
  // Per thread block input and output buffers in shared memory.
	char *inbuf = sdata;
	char *outbuf = sdata + THREADS_PER_BLOCK;

	// Clear the output buffer.  
	outbuf[tid] = 0;
	__syncthreads();
	
  // Note the last thread (globally) we want to have search in this buffer. 
  const unsigned int bytes_for_last_block = bufsize % (THREADS_PER_BLOCK - longestneedle);
  const unsigned int last_thread = ((num_thread_blocks - 1) *\
  		THREADS_PER_BLOCK) + (bytes_for_last_block - 1);
  
  // Get the block of global data that this thread block is responsible for.
  // Must account for needles which might overlap block boundaries.
  inbuf[tid] = dev_in[g_tid - (longestneedle * bid)];
  __syncthreads();

		
	int case_insen = FALSE;	// Is the current needle case insensitive?
	int result_ix = 0;			// Index in output buffer where we can write.
	int wc_shift = 0;				// For needles with leading wildcards.
	int i = 0, j = 0;
	
	// Lets find headers.
	// Loop over potential matches.
	// Get first potential match.
	char pindex = lookup_headers[(unsigned char)inbuf[tid]][i];
	while(pindex != LOOKUP_ENDOFROW) {
		j=0;
		case_insen = !(pattern[pindex][PATTERN_CASESEN]);
		wc_shift = pattern[pindex][PATTERN_WCSHIFT];

		if((((int)tid - wc_shift) >= 0) &&  
				(tid < (THREADS_PER_BLOCK - longestneedle + wc_shift)) && 
				(g_tid < last_thread + wc_shift)){
			// This may look complicated, but breaking it up is Retardedly Slower. 
			// while(inbuf matches pattern) {
			while(((inbuf[tid +j] == pattern[pindex][j+1+wc_shift]) ||	// direct match
					(pattern[pindex][j+1+wc_shift] == wildcard) ||					// wildcard match
					(case_insen && ISALPHA(inbuf[tid+j]) && ISALPHA(pattern[pindex][j+1+wc_shift]) && // case insensitive
					(((inbuf[tid+j] - pattern[pindex][j+1+wc_shift]) == 32) || 
					((inbuf[tid+j] - pattern[pindex][j+1+wc_shift]) == -32))))  &&
					j < (pattern[pindex][0] - wc_shift)) {
				j++;
    	}
     	// Have match if number of matching bytes equals the size of the pattern.
     	if (j == (pattern[pindex][0] - wc_shift)) {
				// Result_ix keeps track of where in the output buffer we can write.
				// atomicAdd returns the pre-incremented value. We encode a match as
				// 2 bytes, 1st: the matching pattern index, 2nd: the position in the
				// input buffer where the match begins.
				result_ix = atomicAdd(&dev_count[bid], 1);
				outbuf[result_ix*2] = pindex/2 + 1;
				outbuf[(result_ix*2)+1] = tid - wc_shift;	
     	}
    }
		i+=1;
		// Get next potential match.
		pindex = lookup_headers[(unsigned char)inbuf[tid]][i];
	}
	__syncthreads();
		
	// Same as above but for footers. No shift as we assume footers will not
	// have leading wildcard characters.
	if ((tid < (THREADS_PER_BLOCK - longestneedle)) && (g_tid < last_thread)){
	i=0; j=0;
	pindex = lookup_footers[(unsigned char)inbuf[tid]][i];
	while(pindex != LOOKUP_ENDOFROW) {
		j=0;
		case_insen = !(pattern[pindex][PATTERN_CASESEN]);
		while(((inbuf[tid + j] == pattern[pindex][j+1]) ||	// direct match
				(pattern[pindex][j+1] == wildcard) ||					// wildcard match
				(case_insen && ISALPHA(inbuf[tid + j]) && ISALPHA(pattern[pindex][j+1]) && // case insensitive
				(((inbuf[tid+j] - pattern[pindex][j+1]) == 32) || 
				((inbuf[tid+j] - pattern[pindex][j+1]) == -32))))  &&
				j < pattern[pindex][0]) {
			j++;
    }
     	if (j == pattern[pindex][0]) {
				result_ix = atomicAdd(&dev_count[bid], 1);
				outbuf[result_ix*2] =  -1 * (pindex/2+1);
				outbuf[(result_ix*2)+1] = tid;
      } 
			i+=1;
			pindex = lookup_footers[(unsigned char)inbuf[tid]][i];
		}
	}
	__syncthreads();


	// The scalpel engine requires that it see the needles in the order in which
	// they appear in the image, the above code gives no ordering guarantees, 
	// so we'll just go ahead and sort, but only if we found any results.
	if(dev_count[bid] > 1) {
		// Sort outbuf by foundat index first. Bubble! Really? Not parallel? Luser.
		if(tid == 0) {
			int changed = TRUE;
			char tmp = 0;
			while(changed) {
				changed = FALSE;
				for(i=0; i<(dev_count[bid]*2) - 2; i+=2) {
					if((unsigned char)outbuf[i+1] > (unsigned char)outbuf[i+3]) {
						tmp = outbuf[i+1];
						outbuf[i+1] = outbuf[i+3];
						outbuf[i+3] = tmp;
						tmp = outbuf[i];
						outbuf[i] = outbuf[i+2];
						outbuf[i+2] = tmp;
						changed = TRUE;
					}
				}
			}
		}
	}
	__syncthreads();				

	// Copy the results buffer out to global RAM, but only if dev_count[bid] says
	// that we found some results to copy back, otherwise skip the write.
	if(dev_count[bid] > 0) {
		if(tid < RESULTS_SIZE_PER_BLOCK) {
			dev_out[(bid*RESULTS_SIZE_PER_BLOCK)+tid] = outbuf[tid];
		}
	}
	__syncthreads();
	// Done with this buffer.
}
	

// Host-side code to set up for the GPU powered search above.
int gpuSearchBuffer(char *input, int bufsize, char *output, int longestneedle, char wildcard) {
	
	int maxblocks = (SIZE_OF_BUFFER / (THREADS_PER_BLOCK - longestneedle))+1;
	
	// copy host buffer to device memory
  CUDA_SAFE_CALL(hipMemcpy(dev_in, input, bufsize,
			    hipMemcpyHostToDevice));
		
	// Set up kernel execution parameters.	
	int num_thread_blocks = (bufsize / (THREADS_PER_BLOCK - longestneedle)) +1;
  dim3 grid(num_thread_blocks, 1, 1);
  dim3 threads(THREADS_PER_BLOCK, 1, 1);
  
  // Clear the results buffer between invocations. Device RAM is persistent
  // across kernel invocations.
  CUDA_SAFE_CALL(hipMemset((void *)dev_count, 0, num_thread_blocks*sizeof(int)));	
  CUDA_SAFE_CALL(hipMemset((void *)dev_out, 0, maxblocks*RESULTS_SIZE_PER_BLOCK));	
  
	// Execute the kernel.
  gpudigbuffer_kernel<<<grid, threads, (THREADS_PER_BLOCK*2)>>>
								 (dev_in, dev_out, bufsize, longestneedle, dev_count, wildcard); 

	// Kernel invocation is asynchronous, wait till done before copying results.
	checkCUDAError("kernel invocation");
  hipDeviceSynchronize();

  // Copy encoded results back to the host.
  CUDA_SAFE_CALL(hipMemcpy(output, dev_out, maxblocks*RESULTS_SIZE_PER_BLOCK, hipMemcpyDeviceToHost));
	
	return GPU_ERROR;
}


// Helper functions.

// Copy the header / footer patterns table to constant memory on the GPU.
void copytodevicepattern(char hostpatterntable[MAX_PATTERNS][MAX_PATTERN_LENGTH]) {
	
	memcpy(host_patterns, hostpatterntable, MAX_PATTERNS*MAX_PATTERN_LENGTH);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pattern), hostpatterntable, sizeof(pattern), 0));
}


// Copy the header lookup table to constant memory on the GPU.
void copytodevicelookup_headers(char hostlookuptable[LOOKUP_ROWS][LOOKUP_COLUMNS]){

	memcpy(hostlookup_headers, hostlookuptable, LOOKUP_ROWS*LOOKUP_COLUMNS);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(lookup_headers), hostlookuptable, sizeof(lookup_headers), 0));
}


// Copy the footer lookup table to constant memory on the GPU.
void copytodevicelookup_footers(char hostlookuptable[LOOKUP_ROWS][LOOKUP_COLUMNS]){

	memcpy(hostlookup_footers, hostlookuptable, LOOKUP_ROWS*LOOKUP_COLUMNS);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(lookup_footers), hostlookuptable, sizeof(lookup_footers), 0));
}


// Wrapper for allocating RAM on the GPU.
void ourCudaMallocHost(void **ptr, int len)
{
		if(hipHostAlloc(ptr, len, hipHostMallocDefault)) {
				fprintf(stderr, "\nERROR: hipHostMalloc \n\n");
		}
}


// Allocate persistent device memory. 
int gpu_init(int longestneedle) {

	// We could set a device here, but to do it intelligently we need to know
	// not only which device is not attached to a display, we also need to know
	// which pci-x bus supports pinned memory. Have no idea how to do that
	// except to test live. Maybe later.

	// Allocate (persistent) device memory for the entire run.
	
	// Input buffer.
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in, SIZE_OF_BUFFER));
	
	// The maximum number of blocks the GPU will run.
	int maxblocks = (SIZE_OF_BUFFER / (THREADS_PER_BLOCK - longestneedle))+1;
	
	// Output buffer.
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out, maxblocks*RESULTS_SIZE_PER_BLOCK));
	
	// Buffer for counts of needles found per block. 
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_count,maxblocks*sizeof(int)));
		
	return GPU_ERROR;
}


// Free persistent device memory.
int gpu_cleanup() {

	CUDA_SAFE_CALL(hipFree(dev_in));	
	CUDA_SAFE_CALL(hipFree(dev_out));
	CUDA_SAFE_CALL(hipFree(dev_count));
	
	return GPU_ERROR;

}


// Checks if the last CUDA operation generated an error.
static void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

/*
// Set us up to use the last device (least likely to be use by display?)
// Pay attention to which device gived speedups for pinned memory, it's
// not necessarily all of them.
static int set_last_device() {

	int device_count;
    CUDA_SAFE_CALL(hipGetDeviceCount(&device_count));
    if(device_count == 0) {
        fprintf(stderr, "There is no device supporting CUDA\n");
	}
	device_count--;
	fprintf(stderr, "Setting device %d\n", device_count); 
	hipSetDevice(device_count);
	
	return GPU_ERROR;
	
}


// Print out info on devices installed on the system
static int enumerate_devices() {
	
	int deviceCount;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");
    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                printf("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
                printf("There is 1 device supporting CUDA\n");
            else
                printf("There are %d devices supporting CUDA\n", deviceCount);
        }
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Major revision number:                         %d\n",
               deviceProp.major);
        printf("  Minor revision number:                         %d\n",
               deviceProp.minor);
        printf("  Total amount of global memory:                 %u bytes\n",
               deviceProp.totalGlobalMem);
    #if CUDART_VERSION >= 2000
        printf("  Number of multiprocessors:                     %d\n",
               deviceProp.multiProcessorCount);
        printf("  Number of cores:                               %d\n",
               8 * deviceProp.multiProcessorCount);
    #endif
        printf("  Total amount of constant memory:               %u bytes\n",
               deviceProp.totalConstMem); 
        printf("  Total amount of shared memory per block:       %u bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %u bytes\n",
               deviceProp.memPitch);
        printf("  Texture alignment:                             %u bytes\n",
               deviceProp.textureAlignment);
        printf("  Clock rate:                                    %.2f GHz\n",
               deviceProp.clockRate * 1e-6f);
    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:                 %s\n",
               deviceProp.deviceOverlap ? "Yes" : "No");
    #endif
	
	}
		
	return GPU_ERROR;	
		
}
*/
